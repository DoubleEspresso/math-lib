#include "hip/hip_runtime.h"
#include "matrix.h"
#include "error.cuh"

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

// common shorthands
#define BX blockIdx.x
#define BY blockIdx.y
#define TX threadIdx.x
#define TY threadIdx.y
#define DX blockDim.x 
#define DY blockDim.y
#define TIDX blockDim.x * blockIdx.x + threadIdx.x
#define TIDY blockDim.y * blockIdx.y + threadIdx.y

template<typename T> __global__ void d_lu(T * a, T * b, int * p, T * tp, const int N);

template __global__ void d_lu(double * a, double * b, int * p, double * tp, const int N);
template __global__ void d_lu(float * a, float * b, int * p, float * tp, const int N);
template __global__ void d_lu(int * a, int * b, int * p, int * tp, const int N);
//template __global__ void d_lu(const char * a, char * b, const int N);

// finds most non-zero element along a column
// accounting for possible permutations of rows and examining
// only those rows > srow
template<typename T>
inline void mostnonzero_col(const T * a, int * P, const int srow, const int scol, T * mx, int * idx, const int nrows, const int ncols)
{
	*mx = -T(INFINITY); *idx = srow;
	for (int r = srow; r < nrows; ++r) {
		T t = a[P[r] * ncols + scol];
		t = (t < 0 ? -t : t);
		if (t >(*mx)) { (*mx) = t; (*idx) = r; }
	}
}

namespace Math
{
	template<typename T>
	void Matrix::lu_gpu(T * a, T * b, int * p, const int N)
	{
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		T * ad; T * bd; T * tp; int * pd;
		
		size_t sz = N * N * sizeof(T);

		// store permutations
		for (int j = 0; j < N; ++j) p[j] = j;

		// partial pivoting step
		for (int r = 0; r < N - 1; ++r) {
			T mxval = 0; int ridx = p[r];
			mostnonzero_col<T>(a, p, r, r, &mxval, &ridx, N, N);

			if (ridx > r) // found a row with an entry "farther" from 0.
			{
				int tmp = p[r];
				p[r] = p[ridx]; p[ridx] = tmp;
			}
		}

		gpuCheckErr(hipMalloc((void**)&ad, sz));
		gpuCheckErr(hipMalloc((void**)&bd, sz));
		gpuCheckErr(hipMalloc((void**)&tp, sz));
		gpuCheckErr(hipMalloc((void**)&pd, N * sizeof(int)));

		gpuCheckErr(hipMemcpy(ad, a, sz, hipMemcpyHostToDevice));
		gpuCheckErr(hipMemcpy(bd, b, sz, hipMemcpyHostToDevice));
		gpuCheckErr(hipMemcpy(tp, b, sz, hipMemcpyHostToDevice));
		gpuCheckErr(hipMemcpy(pd, p, N*sizeof(int), hipMemcpyHostToDevice));

		int threads = 16;
		int bs = (N + threads - 1) / threads;
		int blocks = bs < 1024 ? bs : 1024;

		//printf("..threads(%d), blocks(%d)\n", threads, blocks);
		//printf("..dbg a(%d, %d), b(%d, %d), grid(%d,%d), threads(%d,%d)\n", ra, ca, ca, cb, bx, by, Threads.x, Threads.y);

		hipEventRecord(start);
		d_lu<T> << <blocks, threads >> >(ad, bd, pd, tp, N);
		gpuCheckErr(hipPeekAtLastError());
		gpuCheckErr(hipDeviceSynchronize());
		hipEventRecord(stop);

		gpuCheckErr(hipMemcpy(b, bd, sz, hipMemcpyDeviceToHost));
		hipFree(ad);
		hipFree(bd);
		hipFree(tp);
		hipFree(pd);

		float ms = 0;
		hipEventElapsedTime(&ms, start, stop);
		printf("..N(%d) gpu_lu(%3.1fms)\n", N, ms);
	}

	template<typename T> __global__ void d_lu(T * a, T * b, int * p, T * tp, const int N)
	{
		int tx = TIDX;
		int stride = gridDim.x * blockDim.x;
		
		for (int i = 0; i < N; ++i) {
			int piN = p[i] * N;
			int iN = i*N;

			for (int j = i + tx; j < N; j += stride) {
				T sum = 0; int jN = j * N; T aiNj = a[piN + j];

				for (int k = 0; k < i; ++k) aiNj -= b[iN + k] * tp[jN + k];

				b[iN + j] = aiNj;
				tp[jN + i] = aiNj;
			}
			//__syncthreads();

			T div = b[iN + i]; 
			for (int j = i + 1 + tx; j < N; j += stride) {
				int jN = j*N; T ajNi = a[p[j] * N + i];
				for (int k = 0; k < i; ++k) ajNi  -= b[jN + k] * tp[iN + k];

				b[jN + i] = ajNi / div;
				tp[iN + j] = ajNi / div;
			}
			//__syncthreads();
		}
	}
}